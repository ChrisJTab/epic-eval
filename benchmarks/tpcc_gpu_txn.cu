#include "hip/hip_runtime.h"
//
// Created by Shujian Qian on 2024-04-15.
//

#include <benchmarks/tpcc_gpu_txn.cuh>

#include <hipcub/hipcub.hpp>

#include <benchmarks/tpcc_txn.h>
#include <util_gpu_error_check.cuh>

namespace epic::tpcc {

TpccPackedTxnArrayBuilder::TpccPackedTxnArrayBuilder(uint32_t num_txns)
    : num_txns(num_txns)
{
    gpu_err_check(hipMalloc(&txn_sizes, sizeof(uint32_t) * num_txns));
    hipcub::DeviceScan::InclusiveSum(
        nullptr, temp_storage_bytes, static_cast<uint32_t *>(nullptr), static_cast<uint32_t *>(nullptr), num_txns);
    gpu_err_check(hipMalloc(&temp_storage, temp_storage_bytes));
}

static void __global__ calcTxnParamsSizes(GpuPackedTxnArray src, uint32_t *txn_sizes, uint32_t num_txns)
{
    uint32_t txn_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (txn_id >= num_txns)
    {
        return;
    }

    BaseTxn *base_txn_ptr = src.getTxn(txn_id);
    uint32_t txn_type = base_txn_ptr->txn_type;
    constexpr uint32_t tpcc_txn_sizes[6] = {
        0,
        BaseTxnSize<NewOrderTxnParams<FixedSizeTxn>>::value,
        BaseTxnSize<PaymentTxnParams>::value,
        BaseTxnSize<OrderStatusTxnParams>::value,
        BaseTxnSize<DeliveryTxnParams>::value,
        BaseTxnSize<StockLevelTxnParams>::value
    };
    txn_sizes[txn_id] = tpcc_txn_sizes[txn_type];
}

template <>
void TpccPackedTxnArrayBuilder::buildPackedTxnArrayGpu(PackedTxnArray<TpccTxn> &src, PackedTxnArray<TpccTxnParam> &dest)
{
    constexpr uint32_t block_size = 512;
    uint32_t num_blocks = (num_txns + block_size - 1) / block_size;
    /*
    kernel: compute **payload byte-length** of every Txn in the source array

    Each thread inspects one source TpccTxn, determines which TPCC transaction type it is and 
    therefore how many bytes its parameter struct (TpccTxnParam) will occupy when copied, 
    and writes that size into txn_sizes[i]

    txn_sizes[i] now holds the variable-length payload size for txn i.

    [5, 10, 11, 9, 8]
    */
    calcTxnParamsSizes<<<num_blocks, block_size>>>(GpuPackedTxnArray(src), txn_sizes, src.num_txns);


    /*

    Computes an inclusive prefix sum of those sizes directly on the GPU. 
    The scan output is written starting at dest.index[1]; dest.index[0] remains 0.

    After the scan:
        - dest.index[i] = byte offset of payload (i-1) inside the soon-to-be-packed blob;
        - dest.index[num_txns] = total packed byte length (needed later to allocate/copy the blob).   
    
    [0, 5, 15, 26, 35, 43]
    */
    hipcub::DeviceScan::InclusiveSum(temp_storage, temp_storage_bytes, txn_sizes, dest.index + 1, num_txns);
    gpu_err_check(hipDeviceSynchronize());
}

static void __global__ calcTxnExecPlanSizes(GpuPackedTxnArray src, uint32_t *txn_sizes, uint32_t num_txns)
{
    uint32_t txn_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (txn_id >= num_txns)
    {
        return;
    }

    BaseTxn *base_txn_ptr = src.getTxn(txn_id);
    uint32_t txn_type = base_txn_ptr->txn_type;
    constexpr uint32_t tpcc_txn_sizes[6] = {
        0,
        BaseTxnSize<NewOrderExecPlan<FixedSizeTxn>>::value,
        BaseTxnSize<PaymentTxnExecPlan>::value,
        BaseTxnSize<OrderStatusTxnExecPlan>::value,
        BaseTxnSize<DeliveryTxnExecPlan>::value,
        BaseTxnSize<StockLevelTxnExecPlan>::value
    };
    txn_sizes[txn_id] = tpcc_txn_sizes[txn_type];
}

template <>
void TpccPackedTxnArrayBuilder::buildPackedTxnArrayGpu(PackedTxnArray<TpccTxn> &src, PackedTxnArray<TpccExecPlan> &dest)
{
    constexpr uint32_t block_size = 512;
    uint32_t num_blocks = (num_txns + block_size - 1) / block_size;
    calcTxnExecPlanSizes<<<num_blocks, block_size>>>(GpuPackedTxnArray(src), txn_sizes, src.num_txns);
    hipcub::DeviceScan::InclusiveSum(temp_storage, temp_storage_bytes, txn_sizes, dest.index + 1, num_txns);
    gpu_err_check(hipDeviceSynchronize());
}

}
